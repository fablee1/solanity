#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>

#include <iostream>
#include <ctime>

#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "../config.h"

/* -- Types ----------------------------------------------------------------- */

typedef struct {
    uint8_t** states;
} config;

/* -- Prototypes, Because C++ ----------------------------------------------- */

void            vanity_setup(config& vanity);
void            vanity_run(config& vanity);
void __global__ vanity_scan(uint8_t* state, int* keys_found, int* gpu, int* execution_count);
void __device__ b58enc(uint8_t* b58, const uint8_t* data);

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
    ed25519_set_verbose(true);

    config vanity;
    vanity_setup(vanity);
    vanity_run(vanity);
}

std::string getTimeStr(){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

/* -- Vanity Step Functions ------------------------------------------------- */

void vanity_setup(config &vanity) {
    printf("GPU: Initializing Memory\n");
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);
    vanity.states = (uint8_t **) malloc(gpuCount * sizeof(uint8_t *));

    // Create random states so kernels have access to random generators
    // while running in the GPU.
    for (int i = 0; i < gpuCount; ++i) {
        hipSetDevice(i);

        // Fetch Device Properties
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, i);

        // Calculate Occupancy
        int blockSize       = 0,
            minGridSize     = 0,
            maxActiveBlocks = 0;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

        printf("GPU: %d (%s <%d, %d, %d>) -- W: %d, P: %d, TPB: %d, MTD: (%dx, %dy, %dz), MGS: (%dx, %dy, %dz)\n",
            i,
            device.name,
            blockSize,
            minGridSize,
            maxActiveBlocks,
            device.warpSize,
            device.multiProcessorCount,
            device.maxThreadsPerBlock,
            device.maxThreadsDim[0],
            device.maxThreadsDim[1],
            device.maxThreadsDim[2],
            device.maxGridSize[0],
            device.maxGridSize[1],
            device.maxGridSize[2]
        );

        unsigned int n = maxActiveBlocks * blockSize * 32;
        uint8_t *rseed = (uint8_t *) malloc(n);

        std::random_device rd;
        std::uniform_int_distribution<int> dist(0, 255);
        for (unsigned int j = 0; j < n; ++j) {
            rseed[j] = static_cast<uint8_t>(dist(rd));
        }

        hipMalloc((void **)&(vanity.states[i]), n);
        hipMemcpy(vanity.states[i], rseed, n, hipMemcpyHostToDevice); 
    }

    printf("END: Initializing Memory\n");
}

void vanity_run(config &vanity) {
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

    uint64_t executions_total = 0; 
    uint64_t executions_this_iteration; 
    int  executions_this_gpu; 
    int* dev_executions_this_gpu[100];

    int  keys_found_total = 0;
    int  keys_found_this_iteration;
    int* dev_keys_found[100]; // not more than 100 GPUs ok!

    for (int i = 0; i < MAX_ITERATIONS; ++i) {
        auto start = std::chrono::high_resolution_clock::now();

        executions_this_iteration=0;

        // Run on all GPUs
        for (int g = 0; g < gpuCount; ++g) {
            hipSetDevice(g);
            // Calculate Occupancy
            int blockSize       = 0,
                minGridSize     = 0,
                maxActiveBlocks = 0;
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

            int* dev_g;
            hipMalloc((void**)&dev_g, sizeof(int));
            hipMemcpy(dev_g, &g, sizeof(int), hipMemcpyHostToDevice); 

            hipMalloc((void**)&dev_keys_found[g], sizeof(int));        
            hipMalloc((void**)&dev_executions_this_gpu[g], sizeof(int));       

            vanity_scan<<<maxActiveBlocks, blockSize>>>(vanity.states[g], dev_keys_found[g], dev_g, dev_executions_this_gpu[g]);
        }

        hipDeviceSynchronize();
        auto finish = std::chrono::high_resolution_clock::now();

        for (int g = 0; g < gpuCount; ++g) {
            hipMemcpy( &keys_found_this_iteration, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost ); 
            keys_found_total += keys_found_this_iteration; 

            hipMemcpy( &executions_this_gpu, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost ); 
            executions_this_iteration += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
            executions_total += executions_this_gpu * ATTEMPTS_PER_EXECUTION; 
        }

        // Print out performance Summary
        std::chrono::duration<double> elapsed = finish - start;
        printf("%s Iteration %d Attempts: %llu in %f at %fcps - Total Attempts %llu - keys found %d\n",
            getTimeStr().c_str(),
            i + 1,
            (unsigned long long int) executions_this_iteration,
            elapsed.count(),
            executions_this_iteration / elapsed.count(),
            (unsigned long long int) executions_total,
            keys_found_total
        );

        if ( keys_found_total >= STOP_AFTER_KEYS_FOUND ) {
            printf("Enough keys found, Done! \n");
            exit(0);    
        }   
    }

    printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

void __global__ vanity_scan(uint8_t* state, int* keys_found, int* gpu, int* exec_count) {
    int id = threadIdx.x + blockIdx.x * blockDim.x +
        (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x +
        (threadIdx.z + blockIdx.z * blockDim.z) * gridDim.x * blockDim.x * gridDim.y * blockDim.y;

    atomicAdd(exec_count, 1);

    int suffix_letter_counts[MAX_PATTERNS];
    for (unsigned int n = 0; n < sizeof(suffixes) / sizeof(suffixes[0]); ++n) {
        if (MAX_PATTERNS == n) {
            printf("NEVER SPEAK TO ME OR MY SON AGAIN");
            return;
        }
        int letter_count = 0;
        for (; suffixes[n][letter_count] != 0; letter_count++);
        suffix_letter_counts[n] = letter_count;
    }
    uint8_t suffix_ignore_case_char_masks[64];
    for (int i = 0; i < 64; i++) suffix_ignore_case_char_masks[i] = 0xff;
    for (int i = 0; suffix_ignore_case_mask[i] != 0 && i < 64; i++) {
        suffix_ignore_case_char_masks[i] ^= (suffix_ignore_case_mask[i] == '@') << 5;
    }
    
    // Local Kernel State
    ge_p3 A;
    uint32_t seed_limbs[8] = {0};
    uint8_t seed[32] = {0};
    uint8_t publick[32] = {0};
    uint8_t privatek[64] = {0};
    uint8_t key[256] = {0};

    memcpy(seed_limbs, state + id * 32, 32);

    sha512_context md;

    // Optimization approach:
    // Focus optimizing anything within the ATTEMPTS_PER_EXECUTION loop.
    // Make most-likely path be as branchless as possible to minimize wrap divergence.

    for (int attempts = 0; attempts < ATTEMPTS_PER_EXECUTION; ++attempts) {
        memcpy(seed, seed_limbs, 32);
        // sha512_init Inlined
        md.state[0] = UINT64_C(0x6a09e667f3bcc908);
        md.state[1] = UINT64_C(0xbb67ae8584caa73b);
        md.state[2] = UINT64_C(0x3c6ef372fe94f82b);
        md.state[3] = UINT64_C(0xa54ff53a5f1d36f1);
        md.state[4] = UINT64_C(0x510e527fade682d1);
        md.state[5] = UINT64_C(0x9b05688c2b3e6c1f);
        md.state[6] = UINT64_C(0x1f83d9abfb41bd6b);
        md.state[7] = UINT64_C(0x5be0cd19137e2179);

        // sha512_update inlined
        // 
        // All `if` statements from this function are eliminated if we
        // will only ever hash a 32 byte seed input. So inlining this
        // has a drastic speed improvement on GPUs.
        //
        // This means:
        //   * Normally we iterate for each 128 bytes of input, but we are always < 128. So no iteration.
        //   * We can eliminate a MIN(inlen, (128 - md.curlen)) comparison, specialize to 32, branch prediction improvement.
        //   * We can eliminate the in/inlen tracking as we will never subtract while under 128
        //   * As a result, the only thing update does is copy the bytes into the buffer.
        memcpy(md.buf, seed, 32);

        // sha512_final inlined
        // 
        // As update was effectively elimiated, the only time we do
        // sha512_compress now is in the finalize function. We can also
        // optimize this:
        //
        // This means:
        //   * We don't need to care about the curlen > 112 check. Eliminating a branch.
        //   * We only need to run one round of sha512_compress, so we can inline it entirely as we don't need to unroll.
        md.length = 32 * UINT64_C(8);
        md.buf[32] = (uint8_t) 0x80;

        #pragma unroll
        for (int i = 33; i < 120; i++) {
            md.buf[i] = (uint8_t) 0;
        }
        md.curlen = 120;

        STORE64H(md.length, md.buf + 120);

        // Inline sha512_compress
        uint64_t S[8], W[80], t0, t1;

        /* Copy state into S */
        #pragma unroll
        for (int i = 0; i < 8; i++) {
            S[i] = md.state[i];
        }

        /* Copy the state into 1024-bits into W[0..15] */
        #pragma unroll
        for (int i = 0; i < 16; i++) {
            LOAD64H(W[i], md.buf + (8*i));
        }

        /* Fill W[16..79] */
        #pragma unroll
        for (int i = 16; i < 80; i++) {
            W[i] = Gamma1(W[i - 2]) + W[i - 7] + Gamma0(W[i - 15]) + W[i - 16];
        }

        /* Compress */
        #define RND(a,b,c,d,e,f,g,h,i) \
        t0 = h + Sigma1(e) + Ch(e, f, g) + K[i] + W[i]; \
        t1 = Sigma0(a) + Maj(a, b, c);\
        d += t0; \
        h  = t0 + t1;

        #pragma unroll
        for (int i = 0; i < 80; i += 8) {
            RND(S[0],S[1],S[2],S[3],S[4],S[5],S[6],S[7],i+0);
            RND(S[7],S[0],S[1],S[2],S[3],S[4],S[5],S[6],i+1);
            RND(S[6],S[7],S[0],S[1],S[2],S[3],S[4],S[5],i+2);
            RND(S[5],S[6],S[7],S[0],S[1],S[2],S[3],S[4],i+3);
            RND(S[4],S[5],S[6],S[7],S[0],S[1],S[2],S[3],i+4);
            RND(S[3],S[4],S[5],S[6],S[7],S[0],S[1],S[2],i+5);
            RND(S[2],S[3],S[4],S[5],S[6],S[7],S[0],S[1],i+6);
            RND(S[1],S[2],S[3],S[4],S[5],S[6],S[7],S[0],i+7);
        }

        #undef RND

        /* Feedback */
        #pragma unroll
        for (int i = 0; i < 8; i++) {
            md.state[i] = md.state[i] + S[i];
        }

        // We can now output our finalized bytes into the output buffer.
        #pragma unroll
        for (int i = 0; i < 8; i++) {
            STORE64H(md.state[i], privatek+(8*i));
        }

        // ed25519 Hash Clamping
        privatek[0]  &= 248;
        privatek[31] &= 63;
        privatek[31] |= 64;

        // ed25519 curve multiplication to extract a public key.
        ge_scalarmult_base(&A, privatek);
        ge_p3_tobytes(publick, &A);

        b58enc(key, publick);

        #define CONDITIONAL_CASE_CHAR_EQ(a, b, j) ((suffix_ignore_case_char_masks[j] & (a[j] ^ b[j])) == 0)
        #define IN_RANGE_CHAR_EQ(j) { \
            const uint8_t* suffix_start = key + (key_len - suffix_letter_counts[i]); \
            (CONDITIONAL_CASE_CHAR_EQ(suffixes[i], suffix_start, j) | (suffixes[i][j] == '?')); \
        }
        #define CHAR_EQ(j) ((j >= suffix_letter_counts[i]) | IN_RANGE_CHAR_EQ(j))
        #define CHAR4_EQ(k) (CHAR_EQ(k + 0) & CHAR_EQ(k + 1) & CHAR_EQ(k + 2) & CHAR_EQ(k + 3))

        // Get the key length
        int key_len = 0;
        while (key[key_len] && key_len < 256) key_len++;

        for (int i = 0; i < sizeof(suffixes) / sizeof(suffixes[0]); ++i) {
            // Skip if key is too short for this suffix
            if (key_len < suffix_letter_counts[i]) continue;

            // Only do quick check if suffix is long enough and has no wildcards in first 4 chars
            bool do_quick_check = suffix_letter_counts[i] >= 4;
            for (int j = 0; do_quick_check && j < 4; j++) {
                if (suffixes[i][j] == '?') {
                    do_quick_check = false;
                }
            }
            
            // Calculate start position for suffix comparison
            const uint8_t* suffix_start = key + (key_len - suffix_letter_counts[i]);
            
            if (do_quick_check && !(CHAR4_EQ(0))) continue; // Likely path.

            for (int j = 0; j < suffix_letter_counts[i]; ++j) {
                const uint8_t* suffix_start = key + (key_len - suffix_letter_counts[i]);
                if (!((suffix_ignore_case_char_masks[j] & (suffixes[i][j] ^ suffix_start[j])) == 0 || suffixes[i][j] == '?')) break;

                if (j == (suffix_letter_counts[i] - 1)) {
                    atomicAdd(keys_found, 1);

                    printf("GPU %d MATCH %s - ", *gpu, key);
                    for (int n = 0; n < sizeof(seed); n++) { 
                        printf("%02x", (uint8_t) seed[n]); 
                    }
                    printf("\n");
                    
                    printf("[");
                    for (int n = 0; n < sizeof(seed); n++) { 
                        printf("%d,", (uint8_t) seed[n]);
                    }
                    for (int n = 0; n < sizeof(publick); n++) {
                        if (n + 1 == sizeof(publick)) {   
                            printf("%d", publick[n]);
                        } else {
                            printf("%d,", publick[n]);
                        }
                    }
                    printf("]\n");

                    break;
                }
            }
        }

        // Increment Seed.
        seed_limbs[0] += 1;
        seed_limbs[1] += 3;
        seed_limbs[2] += 7;
        seed_limbs[3] += 11;
    }

    // Copy Random State so that future calls of this kernel/thread/block
    // don't repeat their sequences.
    memcpy(state + id * 32, seed_limbs, 32);
}

// Modified from https://github.com/firedancer-io/firedancer/tree/main/src/ballet/base58
void __device__ b58enc(
    uint8_t *b58,
    const uint8_t *data
) {
    #define BINARY_SZ 8
    #define INTERMEDIATE_SZ 9    

    uint32_t binary[BINARY_SZ];
    memcpy(binary, data, 32);

    #pragma unroll
    for (int i = 0; i < 8; ++i) {
        binary[i] = __byte_perm(binary[i], 0, 0x0123);
    }
    
    #define R1_DIV 656356768UL
    #define RAW58_SZ (INTERMEDIATE_SZ * 5)
    
    uint32_t in_leading_0s = (__clz(binary[0]) >> 3) + (binary[0] == 0) * (__clz(binary[1]) >> 3);
    if (in_leading_0s == 8) {
        // Unlikely. Adding this printf somehow improves performance.
        printf("GPU: In leading zeros exceeded\n");
        for (; in_leading_0s < 32; in_leading_0s++) if (data[in_leading_0s]) break;    
    }
    
    uint64_t intermediate[INTERMEDIATE_SZ] = {0};
    
    intermediate[1] += (uint64_t) binary[0] * (uint64_t) 513735UL;
    intermediate[2] += (uint64_t) binary[0] * (uint64_t) 77223048UL;
    intermediate[3] += (uint64_t) binary[0] * (uint64_t) 437087610UL;
    intermediate[4] += (uint64_t) binary[0] * (uint64_t) 300156666UL;
    intermediate[5] += (uint64_t) binary[0] * (uint64_t) 605448490UL;
    intermediate[6] += (uint64_t) binary[0] * (uint64_t) 214625350UL;
    intermediate[7] += (uint64_t) binary[0] * (uint64_t) 141436834UL;
    intermediate[8] += (uint64_t) binary[0] * (uint64_t) 379377856UL;
    intermediate[2] += (uint64_t) binary[1] * (uint64_t) 78508UL;
    intermediate[3] += (uint64_t) binary[1] * (uint64_t) 646269101UL;
    intermediate[4] += (uint64_t) binary[1] * (uint64_t) 118408823UL;
    intermediate[5] += (uint64_t) binary[1] * (uint64_t) 91512303UL;
    intermediate[6] += (uint64_t) binary[1] * (uint64_t) 209184527UL;
    intermediate[7] += (uint64_t) binary[1] * (uint64_t) 413102373UL;
    intermediate[8] += (uint64_t) binary[1] * (uint64_t) 153715680UL;
    intermediate[3] += (uint64_t) binary[2] * (uint64_t) 11997UL;
    intermediate[4] += (uint64_t) binary[2] * (uint64_t) 486083817UL;
    intermediate[5] += (uint64_t) binary[2] * (uint64_t) 3737691UL;
    intermediate[6] += (uint64_t) binary[2] * (uint64_t) 294005210UL;
    intermediate[7] += (uint64_t) binary[2] * (uint64_t) 247894721UL;
    intermediate[8] += (uint64_t) binary[2] * (uint64_t) 289024608UL;
    intermediate[4] += (uint64_t) binary[3] * (uint64_t) 1833UL;
    intermediate[5] += (uint64_t) binary[3] * (uint64_t) 324463681UL;
    intermediate[6] += (uint64_t) binary[3] * (uint64_t) 385795061UL;
    intermediate[7] += (uint64_t) binary[3] * (uint64_t) 551597588UL;
    intermediate[8] += (uint64_t) binary[3] * (uint64_t) 21339008UL;
    intermediate[5] += (uint64_t) binary[4] * (uint64_t) 280UL;
    intermediate[6] += (uint64_t) binary[4] * (uint64_t) 127692781UL;
    intermediate[7] += (uint64_t) binary[4] * (uint64_t) 389432875UL;
    intermediate[8] += (uint64_t) binary[4] * (uint64_t) 357132832UL;
    intermediate[6] += (uint64_t) binary[5] * (uint64_t) 42UL;
    intermediate[7] += (uint64_t) binary[5] * (uint64_t) 537767569UL;
    intermediate[8] += (uint64_t) binary[5] * (uint64_t) 410450016UL;
    intermediate[7] += (uint64_t) binary[6] * (uint64_t) 6UL;
    intermediate[8] += (uint64_t) binary[6] * (uint64_t) 356826688UL;
    intermediate[8] += (uint64_t) binary[7] * (uint64_t) 1UL;
    
    intermediate[7] += intermediate[8] / R1_DIV;
    intermediate[8] %= R1_DIV;
    intermediate[6] += intermediate[7] / R1_DIV;
    intermediate[7] %= R1_DIV;
    intermediate[5] += intermediate[6] / R1_DIV;
    intermediate[6] %= R1_DIV;
    intermediate[4] += intermediate[5] / R1_DIV;
    intermediate[5] %= R1_DIV;
    intermediate[3] += intermediate[4] / R1_DIV;
    intermediate[4] %= R1_DIV;
    intermediate[2] += intermediate[3] / R1_DIV;
    intermediate[3] %= R1_DIV;
    intermediate[1] += intermediate[2] / R1_DIV;
    intermediate[2] %= R1_DIV;
    intermediate[0] += intermediate[1] / R1_DIV;
    intermediate[1] %= R1_DIV;

    uint8_t raw_base58[RAW58_SZ];

    #pragma unroll
    for (int i = 0; i < INTERMEDIATE_SZ; ++i) {
        raw_base58[5 * i + 4] = ((((uint32_t) intermediate[i]) / 1U       ) % 58U);
        raw_base58[5 * i + 3] = ((((uint32_t) intermediate[i]) / 58U      ) % 58U);
        raw_base58[5 * i + 2] = ((((uint32_t) intermediate[i]) / 3364U    ) % 58U);
        raw_base58[5 * i + 1] = ((((uint32_t) intermediate[i]) / 195112U  ) % 58U);
        raw_base58[5 * i + 0] = ( ((uint32_t) intermediate[i]) / 11316496U);    
    }
    memcpy(binary, raw_base58, 8);
    uint32_t raw_leading_0s = (__clz(__byte_perm(binary[0], 0, 0x0123)) >> 3) +
        (binary[0] == 0) * (__clz(__byte_perm(binary[1], 0, 0x0123)) >> 3);

    if (raw_leading_0s == 8) {
        // Unlikely. Adding this printf somehow improves performance.
        printf("GPU: Raw leading zeros exceeded\n");
        for (; raw_leading_0s < RAW58_SZ; raw_leading_0s++) if (raw_base58[raw_leading_0s]) break;    
    }

    uint32_t skip = raw_leading_0s - in_leading_0s;
    
    static uint8_t const b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";
    const uint32_t n = RAW58_SZ - skip;
    #pragma unroll
    for (int i = 0; i < RAW58_SZ; i++) b58[i] = b58digits_ordered[raw_base58[min(skip + i, RAW58_SZ - 1)]];
    b58[n] = 0;
}
